#define __BENCH_GPU_DECONVOLVE_CU__
#include <iostream>
#include <sstream>
#include <vector>
#include <string>
#include <algorithm>

#include "boost/program_options.hpp"
#include "synthetic_data.hpp"
//#include "cpu_nd_fft.hpp"
#include "multiviewnative.h"
//#include "fftw_interface.h"

#include "logging.hpp"

#include <boost/chrono.hpp>
#include <boost/thread.hpp>


#include "gpu_convolve.cuh"
#include "padd_utils.h"
#include "gpu_nd_fft.cuh"
#include "cufft_utils.cuh"

#include "hip/hip_runtime_api.h"


namespace mvn = multiviewnative;
typedef mvn::no_padd<mvn::image_stack> stack_padding;
typedef mvn::inplace_3d_transform_on_device<imageType>
    device_transform;
typedef mvn::gpu_convolve<stack_padding, imageType, unsigned>
    device_convolve;

template <typename Container>
void inplace_gpu_plan_many_fold(std::vector<Container>& _data, int device){

    std::vector<mvn::image_stack> forwarded_kernels(_data.size());
    std::vector<int> reshaped;
    
      for (int v = 0; v < _data.size(); ++v) {

    stack_padding local_padding(&_data[v].stack_shape_[0],
				      &_data[v].kernel_shape_[0]);

    forwarded_kernels[v].resize(local_padding.extents_);
    local_padding.wrapped_insert_at_offsets(_data[v].kernel_, forwarded_kernels[v]);

    //prepare for fft
    reshaped = multiviewnative::gpu::cufft_r2c_shape(forwarded_kernels[v].shape(),forwarded_kernels[v].shape() + 3);
    forwarded_kernels[v].resize(reshaped);
    HANDLE_ERROR(hipHostRegister((void*)forwarded_kernels[v].data(), 
				    forwarded_kernels[v].num_elements()*sizeof(float),
				    hipHostRegisterPortable));
  }

      //unsigned long reshaped_buffer_byte = forwarded_kernels[0].num_elements()*sizeof(float);
  
      std::vector<device_convolve*> image_folds(_data.size(),0);
      
      for (int v = 0; v < _data.size(); ++v) {
	image_folds[v] = new device_convolve(_data[v].stack_.data(),
					     &(_data[v].stack_shape_[0]),
					     &(_data[v].kernel_shape_[0])
					     );
	
      }

      std::vector<float> image_buffer(_data.size()*forwarded_kernels[0].num_elements());
      std::vector<float> kernel_buffer(image_buffer.size());

      for (int v = 0; v < _data.size(); ++v) {
	std::copy(image_folds[v]->padded_image_->data(),
		  image_folds[v]->padded_image_->data() + image_folds[v]->padded_image_->num_elements(),
		  &image_buffer[0] + (v*image_folds[v]->padded_image_->num_elements()));
	std::copy(forwarded_kernels[v].data(),
		  forwarded_kernels[v].data() + forwarded_kernels[v].num_elements(),
		  &kernel_buffer[0] + (v*forwarded_kernels[v].num_elements()));
      }

      //create plan
      std::vector<int> fftready_shape( forwarded_kernels[0].shape(), 
				       forwarded_kernels[0].shape() + 3);
      unsigned fft_size_in_byte_ = sizeof(float)*std::accumulate(fftready_shape.begin(), fftready_shape.end(),1,std::multiplies<int>());

      std::vector<int> fft_shape(_data[0].stack_shape_.begin(), 
				 _data[0].stack_shape_.end());

      
      // std::vector<int> fftready_shape_as_cufftcomplex(fftready_shape.begin(),
      // 						      fftready_shape.end());
      // for ( int & i : fftready_shape_as_cufftcomplex )
      // 	i/=2;

      
      std::vector<int> iembed(fft_shape.begin(),
			      fft_shape.end());
      std::vector<int> oembed(iembed);
      for ( int & i : oembed )
      	i/=2;


      std::copy(fft_shape.rbegin(), fft_shape.rend(),fft_shape.begin());
      
      hipfftHandle* image_plan = new hipfftHandle;
      hipfftHandle* kernel_plan = new hipfftHandle;
      
      HANDLE_CUFFT_ERROR(hipfftPlanMany(image_plan,  //plan
				       3, //rank
				       (int*)&fft_shape[0], //n
				       (int*)&iembed[0],//inembed
				       1, //istride
				       fft_size_in_byte_/sizeof(hipfftReal),//idist
				       (int*)&oembed[0],//onembed
				       1, //ostride
				       fft_size_in_byte_/sizeof(hipfftComplex),//odist
				       HIPFFT_R2C,
				       _data.size()
				       ));

      HANDLE_CUFFT_ERROR(hipfftPlanMany(kernel_plan,  //plan
				       3, //rank
				       (int*)&fft_shape[0], //n
				       (int*)&iembed[0],//inembed
				       1, //istride
				       fft_size_in_byte_/sizeof(hipfftReal),//idist
				       (int*)&oembed[0],//onembed
				       1, //ostride
				       fft_size_in_byte_/sizeof(hipfftComplex),//odist
				       HIPFFT_R2C,
				       _data.size()
				       ));

      
      //alloc on device
      float* d_images = 0;
      HANDLE_ERROR(hipMalloc((void**)&(d_images), _data.size()*fft_size_in_byte_));
      
      float* d_kernels = 0;
      HANDLE_ERROR(hipMalloc((void**)&(d_kernels), _data.size()*fft_size_in_byte_));
      
      //perform convolution
      std::vector<hipStream_t*> streams(2);
      for( unsigned count = 0;count < streams.size();++count ){
	streams[count] = new hipStream_t;
	HANDLE_ERROR(hipStreamCreate(streams[count]));
      }
  
      //transfer to device
      HANDLE_ERROR(hipMemcpyAsync(d_images,
				   &image_buffer[0], 
				   _data.size()*fft_size_in_byte_,
				   hipMemcpyHostToDevice,
				   *streams[0]
				   ));
      
      HANDLE_ERROR(hipMemcpyAsync(d_kernels,
			      &kernel_buffer[0], 
			      _data.size()*fft_size_in_byte_,
			      hipMemcpyHostToDevice,
			      *streams[1]
			      ));

       HANDLE_CUFFT_ERROR(hipfftSetStream(*image_plan,                            
					 *streams[0] )
			  );
       HANDLE_CUFFT_ERROR(hipfftSetStream(*kernel_plan,                            
					 *streams[1] )
			  );
      //transform forward
      HANDLE_CUFFT_ERROR(
			 hipfftExecR2C(*image_plan, d_images, (hipfftComplex*)d_images));
      
      HANDLE_CUFFT_ERROR(
			 hipfftExecR2C(*kernel_plan, d_kernels, (hipfftComplex*)d_kernels));

      HANDLE_ERROR(hipDeviceSynchronize());
      
      
      //multiply
      unsigned eff_fft_num_elements = fft_size_in_byte_/(2*sizeof(float));
      unsigned numThreads = 256;
      unsigned numBlocks = largestDivisor(eff_fft_num_elements, 
					  numThreads);
      
      float scale = 1.0 / float(std::accumulate(_data[0].stack_shape_.begin(),
						_data[0].stack_shape_.end(),
						1,
						std::multiplies<unsigned>()));
      
      for( unsigned v = 0;v < _data.size();++v ){
	modulateAndNormalize_kernel << <numBlocks, numThreads>>>
	  ((hipfftComplex*)d_images  + (v*eff_fft_num_elements), 
	   (hipfftComplex*)d_kernels + (v*eff_fft_num_elements),
	   eff_fft_num_elements, 
	   scale);
	HANDLE_ERROR(hipPeekAtLastError());
      }

      
  //destroy old plan(s)
      HANDLE_CUFFT_ERROR(hipfftDestroy(*kernel_plan));
      delete kernel_plan;

      HANDLE_CUFFT_ERROR(hipfftDestroy(*image_plan));
      HANDLE_CUFFT_ERROR(hipfftPlanMany(image_plan,  //plan
				       3, //n
				       (int*)&oembed[0], //
				       (int*)&oembed[0],//inembed
				       1, //istride
				       fft_size_in_byte_/sizeof(hipfftComplex),//idist
				       (int*)&iembed[0],//onembed
				       1, //ostride
				       fft_size_in_byte_/sizeof(hipfftReal),//odist
				       HIPFFT_C2R,
				       _data.size()
				       ));

      HANDLE_CUFFT_ERROR(hipfftSetStream(*image_plan,                            
					 *streams[0] )
			  );
       
  //transform back
      HANDLE_CUFFT_ERROR(
			 hipfftExecC2R(*image_plan, 
				      (hipfftComplex*)d_images, 
				      d_images));

  


      //transfer back
      HANDLE_ERROR(hipMemcpyAsync(&image_buffer[0],
				   d_images, 
				   _data.size()*fft_size_in_byte_,
				   hipMemcpyDeviceToHost,
				  *streams[0] 
			      ));

    HANDLE_ERROR(hipDeviceSynchronize());

  
      //copy in and resize to input shape
      for ( unsigned i = 0;i<_data.size();++i){
	_data[i].stack_.resize(reshaped);
	std::copy(&image_buffer[0] + (i*_data[i].stack_.num_elements()),
		  &image_buffer[0] + ((i+1)*_data[i].stack_.num_elements()),
		  _data[i].stack_.data());
	_data[i].stack_.resize(_data[i].stack_shape_);
	
      } 
      
      //clean-up
      HANDLE_CUFFT_ERROR(hipfftDestroy(*image_plan));
      delete image_plan;

      for (int v = 0; v < _data.size(); ++v) {
	HANDLE_ERROR(hipHostUnregister((void*)forwarded_kernels[v].data()));
	delete image_folds[v];
      }
      
      for(hipStream_t* s : streams)
	HANDLE_ERROR(hipStreamDestroy(*s));
      
      HANDLE_ERROR(hipFree(d_images));
      HANDLE_ERROR(hipFree(d_kernels));
  
}

template <typename Container>
void inplace_gpu_batched_fold(std::vector<Container>& _data){
  
  std::vector<mvn::image_stack> forwarded_kernels(_data.size());

  std::vector<int> reshaped;


  for (int v = 0; v < _data.size(); ++v) {

    stack_padding local_padding(&_data[v].stack_shape_[0],
				      &_data[v].kernel_shape_[0]);

    forwarded_kernels[v].resize(local_padding.extents_);
    local_padding.wrapped_insert_at_offsets(_data[v].kernel_, forwarded_kernels[v]);

    //prepare for fft
    reshaped = multiviewnative::gpu::cufft_r2c_shape(forwarded_kernels[v].shape(),forwarded_kernels[v].shape() + 3);
    forwarded_kernels[v].resize(reshaped);
    HANDLE_ERROR(hipHostRegister((void*)forwarded_kernels[v].data(), 
				    forwarded_kernels[v].num_elements()*sizeof(float),
				    hipHostRegisterPortable));
  }

  unsigned long reshaped_buffer_byte = forwarded_kernels[0].num_elements()*sizeof(float);
  
  std::vector<device_convolve*> image_folds(_data.size(),0);
  for (int v = 0; v < _data.size(); ++v) {
    image_folds[v] = new device_convolve(_data[v].stack_.data(),
				      &(_data[v].stack_shape_[0]),
				      &(_data[v].kernel_shape_[0])
				      );
		     
  }

  //creating the plans
  std::vector<hipfftHandle *> plans(2 //number of copy engines
				   );
  for (unsigned count = 0; count < plans.size(); ++count) {

      plans[count] = new hipfftHandle;
      HANDLE_CUFFT_ERROR(hipfftPlan3d(plans[count],                 //
				     (int)_data[0].stack_shape_[0], //
				     (int)_data[0].stack_shape_[1], //
				     (int)_data[0].stack_shape_[2], //
				     HIPFFT_R2C)                    //
			 );
      
  }

  //requesting space on device
  std::vector<float*> src_buffers(plans.size(),0);
  for (unsigned count = 0; count < src_buffers.size(); ++count){
    HANDLE_ERROR(hipMalloc((void**)&(src_buffers[count]), reshaped_buffer_byte));
    
  }

  
  //forward all kernels
  batched_fft_async2plans(forwarded_kernels,plans,src_buffers,false);
    
  //perform convolution
  std::vector<hipStream_t*> streams(plans.size());
  for( unsigned count = 0;count < streams.size();++count ){
    streams[count] = new hipStream_t;
    HANDLE_ERROR(hipStreamCreate(streams[count]));
  }
  

  HANDLE_ERROR(hipMemcpyAsync(src_buffers[0],
				 forwarded_kernels[0].data(), 
				 reshaped_buffer_byte,
				 hipMemcpyHostToDevice,
				 *streams[0]
				 ));
  

  for (int v = 0; v < _data.size(); ++v) {

    image_folds[v]->half_inplace<device_transform>(src_buffers[0],src_buffers[1],
						   streams[0], streams[1],
						   v+1 < _data.size() ? forwarded_kernels[v+1].data() : 0);
  }
    
  //clean-up
  for (unsigned count = 0;count < streams.size();++count){
    HANDLE_ERROR(hipStreamSynchronize(*streams[count]));
    HANDLE_ERROR(hipStreamDestroy(*streams[count]));
  }

  for (unsigned count = 0;count < src_buffers.size();++count){
    HANDLE_ERROR(hipFree(src_buffers[count]));
  }

  for (unsigned count = 0;count < plans.size();++count){

    HANDLE_CUFFT_ERROR(hipfftDestroy(*plans[count]));
    delete plans[count];
    plans[count] = 0;
  }


  for (int v = 0; v < _data.size(); ++v) {
    HANDLE_ERROR(hipHostUnregister((void*)forwarded_kernels[v].data()));
    delete image_folds[v];
  }

}

typedef boost::chrono::high_resolution_clock::time_point tp_t;
typedef boost::chrono::milliseconds ms_t;
typedef boost::chrono::nanoseconds ns_t;

namespace po = boost::program_options;

// typedef boost::multi_array<float, 3, fftw_allocator<float> > fftw_image_stack;
// typedef std::vector<float, fftw_allocator<float> > aligned_float_vector;

int main(int argc, char* argv[]) {
  unsigned num_replicas = 8;
  bool verbose = false;
  
  bool plan_many = false;
  int device_id = -1;
  


  int num_repeats = 5;
  std::string stack_dims = "";

  po::options_description desc("Allowed options");

  // clang-format off
  desc.add_options()                                                      //
      ("help,h", "produce help message")                                  //
      ("verbose,v", "print lots of information in between")               //
      ("plan_many,p", "use hipfftPlanMany for transforms")               //
      ("header-only,H", "print header of stats only")                     //
                                                                          //
      ("stack_dimensions,s",                                              //
       po::value<std::string>(&stack_dims)->default_value("64x64x64"),  //
       "HxWxD of synthetic stacks to generate")                           //
                                                                          //
      ("repeats,r",                                                       //
       po::value<int>(&num_repeats)->default_value(10),                   //
       "number of repetitions per measurement")                           //
                                                                          //
      ("num_replicas,n",                                                  //
       po::value<unsigned>(&num_replicas)->default_value(8),              //
       "number of replicas to use for batched processing")                //
                                                                          //
      ("device_id,d",                                                     //
       po::value<int>(&device_id)->default_value(-1),                     //
       "cuda device to use")                                              //
      ;                                                                   //
  // clang-format on

po::variables_map vm;

  po::store(po::parse_command_line(argc, argv, desc), vm);

  po::notify(vm);

  if (vm.count("help")) {
    std::cout << desc << "\n";
    return 1;
  }

  if (vm.count("header-only")) {
    print_header();
    return 0;
  }


  verbose = vm.count("verbose");
  // out_of_place = vm.count("out-of-place");
  plan_many = vm.count("plan_many");

  std::vector<unsigned> numeric_stack_dims;
  split<'x'>(stack_dims, numeric_stack_dims);

  if (verbose) {
    std::cout << "received " << numeric_stack_dims.size() << " dimensions: ";
    for (unsigned i = 0; i < numeric_stack_dims.size(); ++i) {
      std::cout << numeric_stack_dims[i] << " ";
    }
    std::cout << "\n";
  }

  if (numeric_stack_dims.size() != 3) {
    std::cerr << ">> " << numeric_stack_dims.size()
              << "-D data, not supported yet!\n";
    return 1;
  }

  std::vector<unsigned> reshaped(numeric_stack_dims);
  reshaped.back() = (reshaped.back() / 2 + 1) * 2;

  //////////////////////////////////////////////////////////////////////////////
  // set device flags
  if(device_id<0)
    device_id = selectDeviceWithHighestComputeCapability();
  
  HANDLE_ERROR(hipSetDevice(device_id));
  unsigned long cufft_extra_space =
      cufft_3d_estimated_memory_consumption(numeric_stack_dims);
  unsigned long cufft_data_size = multiviewnative::gpu::cufft_r2c_memory(numeric_stack_dims);
  // unsigned long data_size_byte =
  //     std::accumulate(numeric_stack_dims.begin(), numeric_stack_dims.end(), 1u,
  //                     std::multiplies<unsigned long>()) *
  //     sizeof(float);
  unsigned long memory_available_on_device = getAvailableGMemOnCurrentDevice();

  float exp_mem_mb = (cufft_extra_space + cufft_data_size) / float(1 << 20);
  float av_mem_mb = memory_available_on_device / float(1 << 20);

  if (exp_mem_mb > av_mem_mb) {
    std::cerr << "not enough memory available on device, needed " << exp_mem_mb
              << " MB (data only: " << cufft_data_size / float(1 << 20)
              << " MB), available: " << av_mem_mb << " MB\n";
    return 1;
  } else {
    if (verbose)
      std::cout << "cufft memory estimate: needed " << exp_mem_mb
                << " MB (data only: " << cufft_data_size / float(1 << 20)
                << " MB), available: " << av_mem_mb << " MB\n";
  }



  multiviewnative::image_kernel_data raw(numeric_stack_dims);
  multiviewnative::image_kernel_data reference = raw;
  
  inplace_gpu_convolution(reference.stack_.data(),
			  &reference.stack_shape_[0],
			  reference.kernel_.data(),
			  &reference.kernel_shape_[0],
			  device_id);

  std::vector<multiviewnative::image_kernel_data> stacks(num_replicas,raw);


  if (verbose) {
    std::cout << "[config]\t"
              << "\n"
              << "num_replicas\t:\t" << num_replicas << "\nnumeric size\t:\t";
    std::copy(numeric_stack_dims.begin(), numeric_stack_dims.end(),
              std::ostream_iterator<unsigned>(std::cout, " "));

    std::cout << "\nfftw size\t:\t";
    std::copy(reshaped.begin(), reshaped.end(),
              std::ostream_iterator<unsigned>(std::cout, " "));
    std::cout << "\n";
  }


    stacks[0] = raw;

  //start measurement
  std::vector<ns_t> durations(num_repeats);

  ns_t time_ns = ns_t(0);
  tp_t start, end;

  hipProfilerStart();
    
  for (int r = 0; r < num_repeats; ++r) {

    for ( multiviewnative::image_kernel_data& s : stacks ){
      s.stack_ = raw.stack_;
      s.kernel_ = raw.kernel_;
    }
    

    start = boost::chrono::high_resolution_clock::now();

    if(!plan_many)
      inplace_gpu_batched_fold(stacks);
    else
      inplace_gpu_plan_many_fold(stacks, device_id);
    
    end = boost::chrono::high_resolution_clock::now();
    durations[r] = boost::chrono::duration_cast<ns_t>(end - start);

    time_ns += boost::chrono::duration_cast<ns_t>(end - start);
    if (verbose) {
      std::cout << r << "\t"
                << boost::chrono::duration_cast<ns_t>(durations[r]).count() /
	double(1e6) << " ms\n";
    }
  }
  hipProfilerStop();

  bool data_valid = std::equal(reference.stack_.data(), reference.stack_.data() + reference.stack_.num_elements(),
			       stacks[0].stack_.data());
  

  std::string implementation_name = __FILE__;
  std::stringstream comments("");
  comments << "global_plan";
  if(data_valid)
    comments << ",OK";
  else
    comments << ",NA";

  if(plan_many)
    comments << ",plan_many";


  std::string device_name = get_cuda_device_name(device_id);
  std::replace(device_name.begin(), device_name.end(), ' ', '_');

  if(verbose)
    print_header();


  print_info(1,
	     implementation_name,
	     device_name,
	     num_repeats,
	     time_ns.count() / double(1e6),
	     numeric_stack_dims,
	     sizeof(float),
	     comments.str()
	     );


  return 0;
}
