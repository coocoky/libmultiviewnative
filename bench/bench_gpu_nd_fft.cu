#define __BENCH_GPU_DECONVOLVE_CU__
#include <iostream>
#include <sstream>
#include <vector>
#include <string>
#include <sstream>
#include <algorithm>

#include "boost/program_options.hpp"
#include "synthetic_data.hpp"

#include "hip/hip_runtime_api.h"
#include "gpu_nd_fft.cuh"
#include "logging.hpp"

#include <boost/chrono.hpp>
typedef boost::chrono::high_resolution_clock::time_point tp_t;
typedef boost::chrono::milliseconds ms_t;
typedef boost::chrono::nanoseconds ns_t;

// #include <boost/timer/timer.hpp>

// using boost::timer::cpu_timer;
// using boost::timer::cpu_times;
// using boost::timer::nanosecond_type;

namespace po = boost::program_options;

int main(int argc, char* argv[]) {

  bool verbose = false;
  bool with_transfers = false;
  bool with_allocation = false;
  bool out_of_place = false;
  bool use_global_plan = false;
  hipfftHandle* global_plan = 0;

  int num_repeats = 5;
  std::string stack_dims = "";

  po::options_description desc("Allowed options");

  // clang-format off
  desc.add_options() //
    ("help,h", "produce help message") //
    ("header-only,H", "print header of stats only")                   //
    ("verbose,v", "print lots of information in between") //
    ("with_transfers,t", "include host-device transfers in timings")//
    ("global_plan,g","use a global plan, rather than creating a plan everytime a transformation is performed")//
    ("out-of-place,o","perform out-of-place transforms")//
    ("with_allocation,a", "include host-device memory allocation in timings")//

    ("stack_dimensions,s",//
     po::value<std::string>(&stack_dims)->default_value("64x64x64"),//
     "HxWxD of synthetic stacks to generate")//
    
    ("repeats,r", po::value<int>(&num_repeats)->default_value(10),//
     "number of repetitions per measurement")//
    ;//
  //clang-format on

  po::variables_map vm;

  po::store(po::parse_command_line(argc, argv, desc), vm);

  po::notify(vm);

  if (vm.count("help")) {
    std::cout << desc << "\n";
    return 1;
  }

  if (vm.count("header-only")) {
    print_header();
    return 0;
  }

  verbose = vm.count("verbose");
  with_transfers = vm.count("with_transfers");
  with_allocation = vm.count("with_allocation");
  out_of_place = vm.count("out-of-place");
  use_global_plan = vm.count("global_plan");

  std::vector<unsigned> numeric_stack_dims;
  split<'x'>(stack_dims, numeric_stack_dims);

  if (verbose) {
    std::cout << "received " << numeric_stack_dims.size() << " dimensions: ";
    for (unsigned i = 0; i < numeric_stack_dims.size(); ++i) {
      std::cout << numeric_stack_dims[i] << " ";
    }
    std::cout << "\n";
  }

  if (numeric_stack_dims.size() != 3) {
    std::cerr << ">> " << numeric_stack_dims.size()
              << "-D data, not supported yet!\n";
    return 1;
  }

  int device_id = selectDeviceWithHighestComputeCapability();
  HANDLE_ERROR(hipSetDevice(device_id));
  unsigned long cufft_extra_space =
      cufft_3d_estimated_memory_consumption(numeric_stack_dims);
  unsigned long cufft_data_size = multiviewnative::gpu::cufft_r2c_memory(numeric_stack_dims);
  unsigned long data_size_byte =
      std::accumulate(numeric_stack_dims.begin(), numeric_stack_dims.end(), 1u,
                      std::multiplies<unsigned long>()) *
      sizeof(float);
  unsigned long memory_available_on_device = getAvailableGMemOnCurrentDevice();

  float exp_mem_mb = (cufft_extra_space + cufft_data_size) / float(1 << 20);
  float av_mem_mb = memory_available_on_device / float(1 << 20);

  if (exp_mem_mb > av_mem_mb) {
    std::cerr << "not enough memory available on device, needed " << exp_mem_mb
              << " MB (data only: " << cufft_data_size / float(1 << 20)
              << " MB), available: " << av_mem_mb << " MB\n";
    return 1;
  } else {
    if (verbose)
      std::cout << "cufft memory estimate: needed " << exp_mem_mb
                << " MB (data only: " << cufft_data_size / float(1 << 20)
                << " MB), available: " << av_mem_mb << " MB\n";
  }

  if (use_global_plan) {
    global_plan = new hipfftHandle;

    HANDLE_CUFFT_ERROR(hipfftPlan3d(global_plan, (int)numeric_stack_dims[0],
                                   (int)numeric_stack_dims[1],
                                   (int)numeric_stack_dims[2], HIPFFT_R2C));

    HANDLE_CUFFT_ERROR(
        cufftSetCompatibilityMode(*global_plan, CUFFT_COMPATIBILITY_NATIVE));
  }

  multiviewnative::image_kernel_data data(numeric_stack_dims);
  std::random_shuffle(data.stack_.data(),
                      data.stack_.data() + data.stack_.num_elements());
  if (verbose) {
    std::cout << "[config]\t"
              << ((with_allocation) ? "incl_alloc" : "excl_alloc") << " "
              << ((with_transfers) ? "incl_tx" : "excl_tx") << " "
              << ((out_of_place) ? "out-of-place" : "inplace") << " "
              << ((use_global_plan) ? "global plans" : "local plans") << " "
              << "\n";
    data.info();
  }

  std::vector<ns_t> durations(num_repeats);
  tp_t start, end;
  ns_t time_ns = ns_t(0);

  float* d_dest_buffer = 0;
  const unsigned fft_size_in_byte_ = multiviewnative::gpu::cufft_r2c_memory(numeric_stack_dims);
  if (out_of_place)
    HANDLE_ERROR(hipMalloc((void**)&(d_dest_buffer), fft_size_in_byte_));

  if (!with_allocation) {

    float* d_src_buffer = 0;

    if (out_of_place)
      HANDLE_ERROR(hipMalloc((void**)&(d_src_buffer), data_size_byte));
    else
      HANDLE_ERROR(hipMalloc((void**)&(d_src_buffer), fft_size_in_byte_));

    if (with_transfers) {
      // warm-up
      fft_incl_transfer_excl_alloc(data.stack_, d_src_buffer,
                                   out_of_place ? d_dest_buffer : 0,
                                   use_global_plan ? global_plan : 0);

      hipProfilerStart();
      for (int r = 0; r < num_repeats; ++r) {
        start = boost::chrono::high_resolution_clock::now();
        fft_incl_transfer_excl_alloc(data.stack_, d_src_buffer,
                                     out_of_place ? d_dest_buffer : 0,
                                     use_global_plan ? global_plan : 0);

	end = boost::chrono::high_resolution_clock::now();
	durations[r] = boost::chrono::duration_cast<ns_t>(end - start);
	time_ns += durations[r];

        if (verbose) {
          std::cout << r << "\t"
                    << durations[r] / 1e6
                    << " ms\n";
        }
      }
      hipProfilerStop();

    } else {

      unsigned stack_size_in_byte = data.stack_.num_elements() * sizeof(float);
      HANDLE_ERROR(hipHostRegister((void*)data.stack_.data(),
                                    stack_size_in_byte,
                                    hipHostRegisterPortable));
      HANDLE_ERROR(hipMemcpy(d_src_buffer, data.stack_.data(),
                              stack_size_in_byte, hipMemcpyHostToDevice));
      // warm-up
      fft_excl_transfer_excl_alloc(data.stack_, d_src_buffer,
                                   out_of_place ? d_dest_buffer : 0,
                                   use_global_plan ? global_plan : 0);

      hipProfilerStart();
      for (int r = 0; r < num_repeats; ++r) {
        start = boost::chrono::high_resolution_clock::now();
        fft_excl_transfer_excl_alloc(data.stack_, d_src_buffer,
                                     out_of_place ? d_dest_buffer : 0,
                                     use_global_plan ? global_plan : 0);
        end = boost::chrono::high_resolution_clock::now();
	durations[r] = boost::chrono::duration_cast<ns_t>(end - start);
	time_ns += durations[r];

        if (verbose) {
          std::cout << r << "\t"
                    << durations[r] / 1e6
                    << " ms\n";
        }
      }
      hipProfilerStop();

      // to host
      HANDLE_ERROR(hipMemcpy((void*)data.stack_.data(), d_src_buffer,
                              stack_size_in_byte, hipMemcpyDeviceToHost));
      HANDLE_ERROR(hipHostUnregister((void*)data.stack_.data()));
    }

    HANDLE_ERROR(hipFree(d_src_buffer));

  } else {
    with_transfers = true;
    // warm-up
    fft_incl_transfer_incl_alloc(data.stack_, out_of_place ? d_dest_buffer : 0,
                                 use_global_plan ? global_plan : 0);
    // timing should include allocation, which requires including transfers
    hipProfilerStart();
    for (int r = 0; r < num_repeats; ++r) {
      start = boost::chrono::high_resolution_clock::now();
      fft_incl_transfer_incl_alloc(data.stack_,
                                   out_of_place ? d_dest_buffer : 0,
                                   use_global_plan ? global_plan : 0);
      end = boost::chrono::high_resolution_clock::now();
	durations[r] = boost::chrono::duration_cast<ns_t>(end - start);
	time_ns += durations[r];

        if (verbose) {
          std::cout << r << "\t"
                    << durations[r] / 1e6
                    << " ms\n";
        }
    }
    hipProfilerStop();
  }

  if (out_of_place) HANDLE_ERROR(hipFree(d_dest_buffer));

  if (use_global_plan) {
    HANDLE_CUFFT_ERROR(hipfftDestroy(*global_plan));
    delete global_plan;
  }

  
  std::string device_name = get_cuda_device_name(device_id);
  std::replace(device_name.begin(), device_name.end(), ' ', '_');
  
  if(verbose)
    print_header();

  std::stringstream comments;
  comments << ((with_allocation) ? "incl_alloc" : "excl_alloc") << ","
	   << ((with_transfers) ? "incl_tx" : "excl_tx") << ","
	   << ((out_of_place) ? "out-of-place" : "inplace") << ","
	   << ((use_global_plan) ? "global_plan" : "local_plan") ;

  print_info(1,__FILE__,device_name,num_repeats,time_ns.count() / double(1e6),numeric_stack_dims,sizeof(float),comments.str());


  return 0;
}
