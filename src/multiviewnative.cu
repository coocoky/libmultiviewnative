#include "hip/hip_runtime.h"
#define __MULTIVIEWNATIVE_CU__
// ------- C++ ----------
#include <iostream>
#include <cmath>
#include <vector>
#include <algorithm>
#include <numeric>
#include <functional>

// ------- CUDA ----------
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"

// ------- Project ----------
#include "multiviewnative.h"
#include "cuda_helpers.cuh"
#include "cuda_kernels.cuh"
#include "gpu_convolve.cuh"
#include "cufft_utils.cuh"
#include "padd_utils.h"
#include "image_stack_utils.h"

typedef multiviewnative::zero_padd<multiviewnative::image_stack>
    wrap_around_padding;

typedef multiviewnative::zero_padd<multiviewnative::image_stack>
    as_is_padding;

typedef multiviewnative::inplace_3d_transform_on_device<imageType>
    device_transform;

typedef multiviewnative::gpu_convolve<wrap_around_padding, imageType, unsigned>
    device_convolve;

//TODO:
//this is the convolution we wanna use in the end
typedef multiviewnative::gpu_convolve<as_is_padding, imageType, unsigned>
    target_convolve;

/**
   \brief Function to perform an inplace convolution (all inputs will received
   wrap_around_padding)

   \param[in] im 1D array that contains the data image stack
   \param[in] imDim 3D array that contains the shape of the image stack im
   \param[in] kernel 1D array that contains the data kernel stack
   \param[in] kernelDim 3D array that contains the shape of the kernel stack
   kernel
   \param[in] device CUDA device to use (see nvidia-smi for details)

   \return
   \retval

*/
void inplace_gpu_convolution(imageType* im, int* imDim, imageType* kernel,
                             int* kernelDim, int device) {

  using namespace multiviewnative;

  unsigned image_dim[3];
  unsigned kernel_dim[3];
  std::copy(imDim, imDim + 3, &image_dim[0]);
  std::copy(kernelDim, kernelDim + 3, &kernel_dim[0]);

  device_convolve convolver(im, image_dim, kernel, kernel_dim);

  if (device < 0) device = selectDeviceWithHighestComputeCapability();

  convolver.set_device(device);

  convolver.inplace<device_transform>();
}


template <typename padding_type>
void generate_forwarded_kernels(std::vector<multiviewnative::image_stack>& _result,
				workspace input, 
				int kernel_id = 1
				)
{
  if(_result.size()!=input.num_views_)
    _result.resize(input.num_views_);

  int * kernel_dims = 0;
  float * kernel = 0;
  std::vector<int> reshaped;

  for (unsigned v = 0;v < _result.size();++v){
    
    kernel_dims = (kernel_id == 1) ? input.data_[v].kernel1_dims_ : input.data_[v].kernel2_dims_;
    kernel = (kernel_id == 1) ? input.data_[v].kernel1_ : input.data_[v].kernel2_;
    
    multiviewnative::shape_t kernel_shape(kernel_dims, 
					  kernel_dims + multiviewnative::image_stack::dimensionality);
    
    multiviewnative::image_stack_ref kernel_ref(kernel, kernel_shape);
    padding_type padding(input.data_[v].image_dims_, kernel_dims);
    
    //resize to image size
    _result[v].resize(padding.extents_);
    padding.wrapped_insert_at_offsets(kernel_ref, _result[v]);

    //resize to cufft compliance
    reshaped = multiviewnative::gpu::cufft_r2c_shape(_result[v].shape(),_result[v].shape() + 3);
    _result[v].resize(reshaped);
    
    //pin memory
    HANDLE_ERROR(hipHostRegister((void*)_result[v].data(), 
				  _result[v].num_elements()*sizeof(float),
				  hipHostRegisterPortable));
  }

}

/**
   \brief inplace convolution on workspace interlieving host-device copies with
   computations as much as possible
   \details See cuda_memory.cuh for the classes to facilitate this

   \param[in] input workspace that contains all input images, kernels (1+2) and
   weights
   \param[out] psi 3D image stack that will contain the output (it is expected
   to contain some form of start value)
   \param[in] device CUDA device to use (see nvidia-smi for details)

   \return
   \retval

*/
void inplace_gpu_deconvolve_iteration_interleaved(imageType* psi,
                                                  workspace input, int device) {

  using namespace multiviewnative;
  
  const unsigned n_views = input.num_views_;

  std::vector<image_stack> forwarded_kernels1(n_views);
  std::vector<image_stack> forwarded_kernels2(n_views);

  //prepare kernels (padd for cufft)
  generate_forwarded_kernels<as_is_padding>(forwarded_kernels1,input,1);
  generate_forwarded_kernels<as_is_padding>(forwarded_kernels2,input,2);

  std::vector<target_convolve*> view_folds(n_views,0);
  std::vector<image_stack> weights(n_views);

  shape_t input_shape(input.data_[0].image_dims_,input.data_[0].image_dims_ + 3);
  shape_t common_shape(forwarded_kernels1[0].shape(), forwarded_kernels1[0].shape() + 3);
  unsigned long padded_size_byte = forwarded_kernels1[0].num_elements()*sizeof(imageType);

  //prepare image, weights
  for (unsigned v = 0; v < view_folds.size(); ++v) {
    view_folds[v] = new target_convolve(input.data_[v].image_,
					input.data_[v].image_dims_,
					input.data_[v].kernel1_dims_
					);
    weights[v].resize(input_shape);
    std::copy(input.data_[v].weights_, input.data_[v].weights_ + weights[v].num_elements(),
	      weights[v].data());
    weights[v].resize(common_shape);
  }

  //prepare space on device
  std::vector<float*> src_buffers(2);

  for (unsigned count = 0; count < src_buffers.size(); ++count){
    HANDLE_ERROR(hipMalloc((void**)&(src_buffers[count]), padded_size_byte));
  }
  
  
  gpu::batched_fft_async2plans(forwarded_kernels1, common_shape, src_buffers, false);
  gpu::batched_fft_async2plans(forwarded_kernels2, common_shape, src_buffers, false);

  //expand memory on device
  src_buffers.reserve(4);
  for (unsigned count = 0; count < 2; ++count){
    float* temp = 0;
    HANDLE_ERROR(hipMalloc((void**)&(temp), padded_size_byte));
    src_buffers.push_back(temp);
  }

  std::vector<hipStream_t*> streams(2 // TODO: number of copy engines
				     );
  for( unsigned count = 0;count < streams.size();++count ){
    streams[count] = new hipStream_t;
    HANDLE_ERROR(hipStreamCreate(streams[count]));
  }

  //src_buffers is 4 items large
  //use 
  // 0 .. any content 
  // 1 .. any content (mostly kernels)
  // 2 .. integral
  // 3 .. psi
  //fix the indices here
  const int psi_ = 3;
  const int intgr_ = 2;
  

  image_stack_ref input_psi(psi, input_shape);
  image_stack psi_stack = input_psi;
  psi_stack.resize(common_shape);
  
  HANDLE_ERROR(hipMemcpy(src_buffers[psi_],
			  psi_stack.data(), 
			  padded_size_byte,
			  hipMemcpyHostToDevice
			  ));
  
  const unsigned fft_num_elements = forwarded_kernels1[0].num_elements();
  const unsigned eff_fft_num_elements = fft_num_elements / 2;

  unsigned Threads = 128;//optimize later
  unsigned Blocks = largestDivisor(eff_fft_num_elements, Threads);

  for ( int i = 0; i < input.num_iterations_; ++i){

    HANDLE_ERROR(hipMemcpyAsync(src_buffers[1],
				 forwarded_kernels1[0].data(), 
				 padded_size_byte,
				 hipMemcpyHostToDevice,
				 *streams[0]
				 ));

    for (unsigned v = 0; v < n_views; ++v) {

      //integral = psi
      HANDLE_ERROR(hipMemcpy(src_buffers[intgr_],
			      src_buffers[psi_], 
			      padded_size_byte,
			      hipMemcpyDeviceToDevice
			      ));

      //would load internal from host
      // convolve: psi x kernel1 -> psiBlurred :: (Psi*P_v)
      inplace_asynch_convolve_on_device_and_kick<device_transform>(src_buffers[intgr_], 
						 src_buffers[1],
						 &input_shape[0],
						 fft_num_elements,
						 streams,
						 //goes to stream 1, src_buffer 1
						 view_folds[v]->padded_image_->data()
						 );
      
      //get kernel2 into buffer0
      HANDLE_ERROR(hipMemcpyAsync(src_buffers[0],
				 forwarded_kernels2[v].data(), 
				 padded_size_byte,
				 hipMemcpyHostToDevice,
				 *streams[0]
				 ));

      // view / psiBlurred -> psiBlurred :: (phi_v / (Psi*P_v))
      device_divide << <Blocks, Threads, 0, *streams[1] >>>
          (src_buffers[1], src_buffers[intgr_], fft_num_elements);
      HANDLE_LAST_ERROR();

      // convolve: psiBlurred x kernel2 -> integral :: (phi_v / (Psi*P_v)) *
      // P_v^{compound}
      inplace_asynch_convolve_on_device_and_kick<device_transform>(src_buffers[intgr_], 
						 src_buffers[0],
						 &input_shape[0],
						 fft_num_elements,
						 streams,
						 //goes to stream 1, src_buffer 1
						 weights[v].data()
						 );
      
      // computeFinalValues(input_psi,integral,weights)
      // studied impact of different techniques on how to implement this
      // decision (decision in object, decision in if clause)
      // compiler opt & branch prediction seems to suggest this solution
      if (input.lambda_ > 0) {
        device_regularized_final_values <<<Blocks, Threads, 0 , *streams[1]>>>
	  (src_buffers[psi_], src_buffers[intgr_], src_buffers[1],
	   input.lambda_, input.minValue_, fft_num_elements);

      } else {
        device_final_values <<<Blocks, Threads, 0 , *streams[1]>>>
	  (src_buffers[psi_], src_buffers[intgr_], src_buffers[1],
	   input.minValue_, fft_num_elements);
      }
      HANDLE_LAST_ERROR();
	
      //TODO: can this be removed?
      HANDLE_ERROR(hipDeviceSynchronize());
    }

    
  }

  //clean-up
  for (unsigned v = 0; v < n_views; ++v) {
    delete view_folds[v];
  }

  for (unsigned b = 0; b < src_buffers.size(); ++b) {
    HANDLE_ERROR(hipFree(src_buffers[b]));
  }

  //convert all data to what it was
  psi_stack.resize(input_shape);
  
  //copy result
  input_psi = psi_stack;
    
}

/**
   \brief inplace convolution on workspace performing the entire computation on
   device
   \details All data is transferred onto the device first and then the
   computations are performed.
   See cuda_memory.cuh for the classes to facilitate memory transfers.

   \param[in] input workspace that contains all input images, kernels (1+2) and
   weights
   \param[out] psi 3D image stack that will contain the output (it is expected
   to contain some form of start value)
   \param[in] device CUDA device to use (see nvidia-smi for details)

   \return
   \retval

*/
void inplace_gpu_deconvolve_iteration_all_on_device(imageType* psi,
                                                    workspace input,
                                                    int device) {
  HANDLE_ERROR(hipSetDevice(device));

  std::vector<wrap_around_padding> padding(input.num_views_);

  std::vector<multiviewnative::image_stack*> padded_view(input.num_views_);
  std::vector<multiviewnative::image_stack*> padded_kernel1(input.num_views_);
  std::vector<multiviewnative::image_stack*> padded_kernel2(input.num_views_);
  std::vector<multiviewnative::image_stack*> padded_weights(input.num_views_);
  std::vector<size_t> device_memory_elements_required(input.num_views_);

  std::vector<unsigned> image_dim(3);
  std::copy(input.data_[0].image_dims_, input.data_[0].image_dims_ + 3,
            &image_dim[0]);
  std::vector<unsigned> kernel_dim(image_dim.size());
  std::vector<unsigned> cufft_inplace_extents(kernel_dim.size());

  ////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //
  // PREPARE THE DATA (INCL PADDING)
  //
  for (unsigned v = 0; v < input.num_views_; ++v) {

    padding[v] = wrap_around_padding(input.data_[v].image_dims_,
                                     input.data_[v].kernel1_dims_);
    std::copy(input.data_[0].kernel1_dims_, input.data_[0].kernel1_dims_ + 3,
              &kernel_dim[0]);

    padded_view[v] = new multiviewnative::image_stack(padding[v].extents_);
    padded_weights[v] = new multiviewnative::image_stack(padding[v].extents_);
    padded_kernel1[v] = new multiviewnative::image_stack(padding[v].extents_);
    padded_kernel2[v] = new multiviewnative::image_stack(padding[v].extents_);

    multiviewnative::image_stack_cref view(input.data_[v].image_, image_dim);
    multiviewnative::image_stack_cref weights(input.data_[v].weights_,
                                              image_dim);
    multiviewnative::image_stack_cref kernel1(input.data_[v].kernel1_,
                                              kernel_dim);
    multiviewnative::image_stack_cref kernel2(input.data_[v].kernel2_,
                                              kernel_dim);

    padding[v].insert_at_offsets(view, *padded_view[v]);
    padding[v].insert_at_offsets(weights, *padded_weights[v]);
    padding[v].wrapped_insert_at_offsets(kernel1, *padded_kernel1[v]);
    padding[v].wrapped_insert_at_offsets(kernel2, *padded_kernel2[v]);

    multiviewnative::adapt_extents_for_fftw_inplace(
        padding[v].extents_, cufft_inplace_extents,
        padded_view[v]->storage_order());
    device_memory_elements_required[v] = std::accumulate(
        cufft_inplace_extents.begin(), cufft_inplace_extents.end(), 1,
        std::multiplies<size_t>());
  }

  multiviewnative::image_stack_ref input_psi(psi, image_dim);
  multiviewnative::image_stack padded_psi(padding[0].extents_);
  wrap_around_padding input_psi_padder = padding[0];
  input_psi_padder.insert_at_offsets(input_psi, padded_psi);
  unsigned long max_device_memory_elements_required =
      *std::max_element(device_memory_elements_required.begin(),
                        device_memory_elements_required.end());

  dim3 threads(128);
  dim3 blocks(
      largestDivisor(padded_view[0]->num_elements(), size_t(threads.x)));
  ////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //
  // ITERATE
  //
  multiviewnative::stack_on_device<multiviewnative::image_stack> d_running_psi(
      padded_psi, max_device_memory_elements_required);
  multiviewnative::stack_on_device<multiviewnative::image_stack> d_integral(
      max_device_memory_elements_required);
  multiviewnative::stack_on_device<multiviewnative::image_stack> d_view(
      max_device_memory_elements_required);
  multiviewnative::stack_on_device<multiviewnative::image_stack> d_kernel1(
      max_device_memory_elements_required);
  multiviewnative::stack_on_device<multiviewnative::image_stack> d_kernel2(
      max_device_memory_elements_required);
  multiviewnative::stack_on_device<multiviewnative::image_stack> d_weights(
      max_device_memory_elements_required);

  unsigned long long current_gmem_usage_byte =
      6 * max_device_memory_elements_required;
  if (current_gmem_usage_byte > .25 * getAvailableGMemOnCurrentDevice()) {
    std::cout << "current gmem footprint ("
              << current_gmem_usage_byte / float(1 << 20)
              << " MB) exceeds available memory threshold: (free) "
              << getAvailableGMemOnCurrentDevice() / float(1 << 20)
              << " MB, threshold: " << .25 * getAvailableGMemOnCurrentDevice() /
                                           float(1 << 20) << " MB\n";
  }

  for (int iteration = 0; iteration < input.num_iterations_; ++iteration) {

    for (int v = 0; v < input.num_views_; ++v) {

      d_integral = d_running_psi;
      HANDLE_LAST_ERROR();
      d_kernel1.push_to_device(*padded_kernel1[v]);
      HANDLE_LAST_ERROR();
      // integral = integral * kernel1
      multiviewnative::inplace_convolve_on_device<device_transform>(
          d_integral.data(), d_kernel1.data(), &padding[v].extents_[0],
          device_memory_elements_required[v]);
      HANDLE_LAST_ERROR();

      d_view.push_to_device(*padded_view[v]);
      HANDLE_LAST_ERROR();
      device_divide << <blocks, threads>>>
          (d_view.data(), d_integral.data(), padded_view[v]->num_elements());
      HANDLE_LAST_ERROR();
      d_kernel2.push_to_device(*padded_kernel2[v]);
      HANDLE_LAST_ERROR();
      multiviewnative::inplace_convolve_on_device<device_transform>(
          d_integral.data(), d_kernel2.data(), &padding[v].extents_[0],
          device_memory_elements_required[v]);
      HANDLE_LAST_ERROR();
      d_weights.push_to_device(*padded_weights[v]);
      HANDLE_LAST_ERROR();
      if (input.lambda_ > 0) {
        device_regularized_final_values << <blocks, threads>>>
            (d_running_psi.data(), d_integral.data(), d_weights.data(),
             input.lambda_, input.minValue_, padded_view[v]->num_elements());

      } else {
        device_final_values << <blocks, threads>>>
            (d_running_psi.data(), d_integral.data(), d_weights.data(),
             input.minValue_, padded_view[v]->num_elements());
      }
      HANDLE_LAST_ERROR();
    }
  }

  d_running_psi.pull_from_device(padded_psi);

  input_psi = padded_psi
      [boost::indices[multiviewnative::range(
          input_psi_padder.offsets_[0],
          input_psi_padder.offsets_[0] + input_psi.shape()[0])]
                     [multiviewnative::range(
                         input_psi_padder.offsets_[1],
                         input_psi_padder.offsets_[1] + input_psi.shape()[1])]
                     [multiviewnative::range(
                         input_psi_padder.offsets_[2],
                         input_psi_padder.offsets_[2] + input_psi.shape()[2])]];

  ////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //
  // CLEAN-UP
  //
  for (int v = 0; v < input.num_views_; ++v) {

    delete padded_view[v];
    delete padded_kernel1[v];
    delete padded_kernel2[v];
    delete padded_weights[v];
  }
}

/**
   \brief dispatch function, this one decides wether to try and do the entire
   computation on the device or not and then dispatches the appropriate call



   \return
   \retval

*/
void inplace_gpu_deconvolve(imageType* psi, workspace input, int device) {

  if (device < 0) device = selectDeviceWithHighestComputeCapability();


  long long device_gmem_byte = getMemDeviceCUDA(device);
  unsigned device_gmem_mb = device_gmem_byte >> 20;

  size_t cufft_workarea = 0;
  hipfftEstimate3d(input.data_[0].image_dims_[0], input.data_[0].image_dims_[1],
                  input.data_[0].image_dims_[2], HIPFFT_R2C, &cufft_workarea);
  HANDLE_LAST_ERROR();
  float cufft_workarea_mb = cufft_workarea/(1024*1024.);
  
  size_t single_stack_in_byte =
      sizeof(imageType) * std::accumulate(input.data_[0].image_dims_,
                                          input.data_[0].image_dims_ + 3, 1.,
                                          std::multiplies<int>());
  float single_stack_in_mb = single_stack_in_byte / (1024*1024.);

  float memory_fft_step_mb = 2*single_stack_in_mb + cufft_workarea_mb;
  float regularisation_step_mb = 3*single_stack_in_mb;
  float min_memory_budget_mb = std::max(regularisation_step_mb,memory_fft_step_mb);

  // decide if the incoming data fills the memory on device too much
  float memory_all_on_device_mb = (4*input.num_views_ + 2)*single_stack_in_mb + cufft_workarea_mb;
  

  // cufft is memory hungry, that is why we only push all stacks to device mem
  // if the total budget does not exceed 1/3 device mem
  bool all_on_device = memory_all_on_device_mb < (device_gmem_mb * .9);
  std::cout << "[lmvn::inplace_gpu_deconvolve] FFT: "
            << memory_all_on_device_mb << " MB (all-on-device), "
	    << min_memory_budget_mb << " MB (min-mem interleaved), "
            << " available on GPU: " << device_gmem_mb
            << " MB ... ";

  if (all_on_device){
    std::cout << "all on device!\n";
    inplace_gpu_deconvolve_iteration_all_on_device(psi, input, device);
    return;
  }

  if(min_memory_budget_mb < device_gmem_mb){
    std::cout << "interleaved!\n";
    inplace_gpu_deconvolve_iteration_interleaved(psi, input, device);
    return;
  }

  std::cerr << "[lmvn::inplace_gpu_deconvolve] FFT: Unable to run on GPU due to memory constraints!\n";
  
}

#ifndef LB_MAX_THREADS
#define LB_MAX_THREADS 1024
#endif

#ifndef DIMSIMAGE
static const int dimsImage = 3;
#else
static const int dimsImage = DIMSIMAGE;
#endif

__global__ void __launch_bounds__(LB_MAX_THREADS)
    fftShiftKernel(imageType* kernelCUDA, imageType* kernelPaddedCUDA,
                   unsigned int kernelDim_0, unsigned int kernelDim_1,
                   unsigned int kernelDim_2, unsigned int imDim_0,
                   unsigned int imDim_1, unsigned int imDim_2) {
  long int kernelSize = kernelDim_0 * kernelDim_1 * kernelDim_2;
  long int imageSize = imDim_0 * imDim_1 * imDim_2;

  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  long int x, y, z, aux;
  if (tid < kernelSize) {
    // find coordinates
    z = tid - (tid / kernelDim_2) * kernelDim_2;
    aux = (tid - z) / kernelDim_2;
    y = aux - (aux / kernelDim_1) * kernelDim_1;
    x = (aux - y) / kernelDim_1;

    // center coordinates
    x -= (long int)kernelDim_0 / 2;
    y -= (long int)kernelDim_1 / 2;
    z -= (long int)kernelDim_2 / 2;

    // circular shift if necessary
    if (x < 0) x += imDim_0;
    if (y < 0) y += imDim_1;
    if (z < 0) z += imDim_2;

    // WOW! this is a depth-major format
    // calculate position in padded kernel
    aux = z + imDim_2 * (y + imDim_1 * x);

    // copy value
    if (aux < imageSize)
      kernelPaddedCUDA[aux] = kernelCUDA[tid];  // for the most part it should
                                                // be a coalescent access in oth
                                                // places
  }
}

//=====================================================================
// WARNING: for cuFFT the fastest running index is z direction!!! so pos = z +
// imDim[2] * (y + imDim[1] * x)
// NOTE: to avoid transferring a large padded kernel, since memcpy is a limiting
// factor
void convolution3DfftCUDAInPlace(imageType* im, int* imDim, imageType* kernel,
                                 int* kernelDim, int devCUDA) {
  imageType* imCUDA = NULL;
  imageType* kernelCUDA = NULL;

  HANDLE_ERROR(hipSetDevice(devCUDA));

  size_t imSize = std::accumulate(imDim, imDim + 3, 1, std::multiplies<int>());
  size_t kernelSize =
      std::accumulate(kernelDim, kernelDim + 3, 1, std::multiplies<int>());

  size_t imSizeFFT = imSize;
  imSizeFFT +=
      2 * imDim[0] * imDim[1];  // size of the R2C transform in cuFFTComplex
  size_t imSizeFFTInByte = imSizeFFT * sizeof(imageType);
  size_t imSizeInByte = imSize * sizeof(imageType);
  size_t kernelSizeInByte = (kernelSize) * sizeof(imageType);
  // allocat ememory in GPU
  HANDLE_ERROR(hipMalloc((void**)&(imCUDA), imSizeFFTInByte));  // a little bit
                                                                 // larger to
                                                                 // allow
                                                                 // in-place FFT
  HANDLE_ERROR(hipMalloc((void**)&(kernelCUDA), kernelSizeInByte));

  HANDLE_ERROR(
      hipMemcpy(kernelCUDA, kernel, kernelSizeInByte, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(imCUDA, im, imSizeInByte, hipMemcpyHostToDevice));

  ///////////////////////////////////////////////////////////////////////
  convolution3DfftCUDAInPlace_core(imCUDA, imDim, kernelCUDA, kernelDim,
                                   devCUDA);
  ///////////////////////////////////////////////////////////////////////

  // copy result to host and overwrite image
  HANDLE_ERROR(hipMemcpy(im, imCUDA, sizeof(imageType) * imSize,
                          hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipFree(imCUDA));
  HANDLE_ERROR(hipFree(kernelCUDA));
}

//=====================================================================
// WARNING: for cuFFT the fastest running index is z direction!!! so pos = z +
// imDim[2] * (y + imDim[1] * x)
// NOTE: to avoid transferring a large padded kernel, since memcpy is a limiting
// factor
void convolution3DfftCUDAInPlace_core(imageType* _d_imCUDA, int* imDim,
                                      imageType* _d_kernelCUDA, int* kernelDim,
                                      int devCUDA) {
  hipfftHandle fftPlanFwd, fftPlanInv;
  imageType* kernelPaddedCUDA = NULL;

  size_t imSize = 1;
  size_t kernelSize = 1;
  for (int ii = 0; ii < dimsImage; ii++) {
    imSize *= (imDim[ii]);
    kernelSize *= (kernelDim[ii]);
  }

  size_t imSizeFFT = imSize;
  imSizeFFT += 2 * imDim[0] * imDim[1];
  size_t imSizeFFTInByte = imSizeFFT * sizeof(imageType);

  HANDLE_ERROR(hipMalloc((void**)&(kernelPaddedCUDA), imSizeFFTInByte));
  HANDLE_ERROR(hipMemset(kernelPaddedCUDA, 0, imSizeFFTInByte));

  size_t max_threads_on_device = getMaxNThreadsOfDevice(devCUDA);
  size_t max_blocks_in_x =
      getMaxNBlocksOfDevice(devCUDA, 0);  // we are using dim1 blocks only

  int numThreads = std::min(max_threads_on_device, kernelSize);
  size_t numBlocksFromImage = (kernelSize + numThreads - 1) / (numThreads);
  int numBlocks = std::min(max_blocks_in_x, numBlocksFromImage);

  fftShiftKernel << <numBlocks, numThreads>>>
      (_d_kernelCUDA, kernelPaddedCUDA, kernelDim[0], kernelDim[1],
       kernelDim[2], imDim[0], imDim[1], imDim[2]);
  HANDLE_ERROR_KERNEL;

  // make sure GPU finishes
  HANDLE_ERROR(hipDeviceSynchronize());

  hipfftPlan3d(&fftPlanFwd, imDim[0], imDim[1], imDim[2], HIPFFT_R2C);
  HANDLE_ERROR_KERNEL;
  cufftSetCompatibilityMode(fftPlanFwd, CUFFT_COMPATIBILITY_NATIVE);
  HANDLE_ERROR_KERNEL;  // for highest performance since we do not need FFTW
                        // compatibility

  // inPlace FFT for image and kernel
  hipfftExecR2C(fftPlanFwd, _d_imCUDA, (hipfftComplex*)_d_imCUDA);
  HANDLE_ERROR_KERNEL;
  hipfftExecR2C(fftPlanFwd, kernelPaddedCUDA, (hipfftComplex*)kernelPaddedCUDA);
  HANDLE_ERROR_KERNEL;

  size_t halfImSizeFFT = imSizeFFT / 2;
  numThreads = std::min(max_threads_on_device, halfImSizeFFT);
  numBlocksFromImage = (halfImSizeFFT + numThreads - 1) / (numThreads);
  numBlocks = std::min(max_blocks_in_x, numBlocksFromImage);

  // convolve
  float scale = 1.0f / float(imSize);
  modulateAndNormalize_kernel << <numBlocks, numThreads>>>
      ((hipfftComplex*)_d_imCUDA, (hipfftComplex*)kernelPaddedCUDA, halfImSizeFFT,
       scale);
  HANDLE_ERROR_KERNEL;  // last parameter is the size of the FFT

  // inverse FFT of image only
  hipfftPlan3d(&fftPlanInv, imDim[0], imDim[1], imDim[2], HIPFFT_C2R);
  HANDLE_ERROR_KERNEL;
  cufftSetCompatibilityMode(fftPlanInv, CUFFT_COMPATIBILITY_NATIVE);
  HANDLE_ERROR_KERNEL;
  hipfftExecC2R(fftPlanInv, (hipfftComplex*)_d_imCUDA, _d_imCUDA);
  HANDLE_ERROR_KERNEL;

  // release memory
  HANDLE_ERROR(hipFree(kernelPaddedCUDA));
  (hipfftDestroy(fftPlanFwd));
  HANDLE_ERROR_KERNEL;
  (hipfftDestroy(fftPlanInv));
  HANDLE_ERROR_KERNEL;
}

void compute_quotient(imageType* _input, imageType* _output, size_t _size,
                      int _device) {

  imageType* d_input = 0;
  imageType* d_output = 0;

  const size_t sizeInByte = _size * sizeof(imageType);
  HANDLE_ERROR(hipSetDevice(_device));

  HANDLE_ERROR(hipMalloc((void**)&d_input, sizeInByte));
  HANDLE_ERROR(hipMalloc((void**)&d_output, sizeInByte));

  HANDLE_ERROR(hipMemcpy(d_input, _input, sizeInByte, hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(d_output, _output, sizeInByte, hipMemcpyHostToDevice));

  size_t items_per_block = 128;
  size_t items_per_grid = _size / items_per_block;

  dim3 threads(items_per_block);
  dim3 blocks(items_per_grid);

  fit_2Dblocks_to_threads_for_device(threads, blocks, _device);

  // performs d_input_[:]=d_output_[:]/d_input_[:]
  device_divide << <blocks, threads>>> (d_input, d_output, (unsigned int)_size);

  HANDLE_ERROR(
      hipMemcpy(_output, d_output, sizeInByte, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipFree(d_input));
  HANDLE_ERROR(hipFree(d_output));
}

void compute_final_values(imageType* _image, imageType* _integral,
                          imageType* _weight, size_t _size, float _minValue,
                          double _lambda, int _device) {

  imageType* d_integral = 0;
  imageType* d_weight = 0;
  imageType* d_image = 0;

  const size_t sizeInByte = _size * sizeof(imageType);

  HANDLE_ERROR(hipMalloc((void**)&d_integral, sizeInByte));
  HANDLE_ERROR(hipMalloc((void**)&d_weight, sizeInByte));
  HANDLE_ERROR(hipMalloc((void**)&d_image, sizeInByte));

  HANDLE_ERROR(
      hipMemcpy(d_integral, _integral, sizeInByte, hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(d_weight, _weight, sizeInByte, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(d_image, _image, sizeInByte, hipMemcpyHostToDevice));

  size_t items_per_block = 128;
  size_t items_per_grid = _size / items_per_block;

  dim3 threads(items_per_block);
  dim3 blocks(items_per_grid);
  fit_2Dblocks_to_threads_for_device(threads, blocks, _device);

  if (_lambda > 0.)
    device_regularized_final_values << <blocks, threads>>>
        (d_image, d_integral, d_weight, _lambda, _minValue, _size);
  else
    device_final_values << <blocks, threads>>>
        (d_image, d_integral, d_weight, _minValue, _size);

  HANDLE_ERROR(hipMemcpy(_image, d_image, sizeInByte, hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipFree(d_integral));
  HANDLE_ERROR(hipFree(d_weight));
  HANDLE_ERROR(hipFree(d_image));
}

void iterate_fft_plain(imageType* _input, imageType* _kernel,
                       imageType* _output, int* _input_dims, int* _kernel_dims,
                       int _device) {

  size_t inputSize = std::accumulate(
      &_input_dims[0], &_input_dims[0] + dimsImage, 1, multiplies<int>());
  size_t kernelSize = std::accumulate(
      &_kernel_dims[0], &_kernel_dims[0] + dimsImage, 1, multiplies<int>());

  size_t inputInByte = inputSize * sizeof(imageType);
  size_t kernelInByte = kernelSize * sizeof(imageType);

  imageType* input = 0;
  imageType* kernel1 = 0;
  imageType* weights = 0;
  imageType* kernel2 = 0;
  HANDLE_ERROR(
      hipHostMalloc((void**)&(kernel2), kernelInByte, hipHostMallocDefault));
  HANDLE_ERROR(
      hipHostMalloc((void**)&(weights), inputInByte, hipHostMallocDefault));
  HANDLE_ERROR(
      hipHostMalloc((void**)&(kernel1), kernelInByte, hipHostMallocDefault));
  HANDLE_ERROR(
      hipHostMalloc((void**)&(input), inputInByte, hipHostMallocDefault));

  std::fill(&kernel2[0], &kernel2[0] + kernelSize, .1f);
  std::fill(&weights[0], &weights[0] + inputSize, 1.f);
  std::copy(&_kernel[0], &_kernel[0] + kernelSize, &kernel1[0]);
  std::copy(&_input[0], &_input[0] + inputSize, &input[0]);

  //////////////////////////////////////////////////////////////////////////////////////////
  //
  // Entering Loop here
  //
  imageType* d_image_ = 0;
  imageType* d_initial_ = 0;
  imageType* d_kernel_ = 0;
  imageType* d_weights_ = 0;

  size_t imSizeFFT =
      inputSize +
      2 * _input_dims[0] *
          _input_dims[1];  // size of the R2C transform in cuFFTComplex
  size_t imSizeFFTInByte = imSizeFFT * sizeof(imageType);
  int gpu_device = selectDeviceWithHighestComputeCapability();

  HANDLE_ERROR(hipMalloc((void**)&(d_image_), imSizeFFTInByte));  // a little
                                                                   // bit larger
                                                                   // to allow
                                                                   // in-place
                                                                   // FFT
  HANDLE_ERROR(hipMalloc((void**)&(d_initial_), inputInByte));
  HANDLE_ERROR(hipMalloc((void**)&(d_weights_), inputInByte));
  HANDLE_ERROR(hipMalloc((void**)&(d_kernel_), kernelInByte));
  hipStream_t initial_stream, weights_stream;
  HANDLE_ERROR(hipStreamCreate(&initial_stream));
  HANDLE_ERROR(hipStreamCreate(&weights_stream));

  // TODO: should the weights be updated from device_divide (unclear in the java
  // application)?
  HANDLE_ERROR(hipMemcpyAsync(d_weights_, weights, inputInByte,
                               hipMemcpyHostToDevice, weights_stream));
  HANDLE_ERROR(hipMemcpyAsync(d_initial_, input, inputInByte,
                               hipMemcpyHostToDevice, initial_stream));

  HANDLE_ERROR(
      hipMemcpy(d_kernel_, kernel1, kernelInByte, hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(d_image_, input, inputInByte, hipMemcpyHostToDevice));

  // convolve(input) with kernel1 -> psiBlurred
  convolution3DfftCUDAInPlace_core(d_image_, _input_dims, d_kernel_,
                                   _kernel_dims, gpu_device);

  // computeQuotient(psiBlurred,input)
  size_t items_per_block = 128;
  size_t items_per_grid = inputSize / items_per_block;

  dim3 threads(items_per_block);
  dim3 blocks(items_per_grid);
  fit_2Dblocks_to_threads_for_device(threads, blocks, gpu_device);

  // performs d_initial_[:]=d_image_[:]/d_initial_[:]
  // TODO: should the weights be updated?
  device_divide << <blocks, threads, 0, initial_stream>>>
      (d_initial_, d_image_, inputSize);

  // convolve(psiBlurred) with kernel2 -> integral
  HANDLE_ERROR(
      hipMemcpy(d_kernel_, kernel2, kernelInByte, hipMemcpyHostToDevice));
  convolution3DfftCUDAInPlace_core(d_image_, _input_dims, d_kernel_,
                                   _kernel_dims, gpu_device);
  // computeFinalValues(input,integral,weights)
  device_final_values << <blocks, threads, 0, weights_stream>>>
      (d_initial_, d_image_, d_weights_, .0001f, inputSize);

  HANDLE_ERROR(hipMemcpyAsync(_output, d_initial_, inputInByte,
                               hipMemcpyDeviceToHost, weights_stream));
  HANDLE_ERROR(hipStreamSynchronize(weights_stream));

  HANDLE_ERROR(hipFree(d_image_));
  HANDLE_ERROR(hipFree(d_initial_));
  HANDLE_ERROR(hipFree(d_kernel_));
  HANDLE_ERROR(hipFree(d_weights_));

  HANDLE_ERROR(hipHostFree(kernel2));
  HANDLE_ERROR(hipHostFree(weights));
  HANDLE_ERROR(hipHostFree(kernel1));
  HANDLE_ERROR(hipHostFree(input));
  HANDLE_ERROR(hipStreamDestroy(initial_stream));
  HANDLE_ERROR(hipStreamDestroy(weights_stream));
}

void iterate_fft_tikhonov(imageType* _input, imageType* _kernel,
                          imageType* _output, int* _input_dims,
                          int* _kernel_dims, size_t _size, float _minValue,
                          double _lambda, int _device) {

  size_t inputSize = std::accumulate(
      &_input_dims[0], &_input_dims[0] + dimsImage, 1, multiplies<int>());
  size_t kernelSize = std::accumulate(
      &_kernel_dims[0], &_kernel_dims[0] + dimsImage, 1, multiplies<int>());

  size_t inputInByte = inputSize * sizeof(imageType);
  size_t kernelInByte = kernelSize * sizeof(imageType);

  std::vector<imageType>* kernel2_ = new std::vector<imageType>(kernelSize);
  std::vector<imageType>* weights_ = new std::vector<imageType>(inputSize);
  std::fill(kernel2_->begin(), kernel2_->end(), .1f);
  std::fill(weights_->begin(), weights_->end(), 1.f);

  //////////////////////////////////////////////////////////////////////////////////////////
  //
  // Entering Loop here
  //
  imageType* d_image_ = 0;
  imageType* d_initial_ = 0;
  imageType* d_kernel_ = 0;
  imageType* d_weights_ = 0;

  size_t imSizeFFT =
      inputSize +
      2 * _input_dims[0] *
          _input_dims[1];  // size of the R2C transform in cuFFTComplex
  size_t imSizeFFTInByte = imSizeFFT * sizeof(imageType);
  int gpu_device = selectDeviceWithHighestComputeCapability();

  HANDLE_ERROR(hipMalloc((void**)&(d_image_), imSizeFFTInByte));  // a little
                                                                   // bit larger
                                                                   // to allow
                                                                   // in-place
                                                                   // FFT
  HANDLE_ERROR(hipMalloc((void**)&(d_initial_), inputInByte));
  HANDLE_ERROR(hipMalloc((void**)&(d_weights_), inputInByte));
  HANDLE_ERROR(hipMalloc((void**)&(d_kernel_), kernelInByte));
  HANDLE_ERROR(hipMemcpy(d_weights_, &weights_[0], inputInByte,
                          hipMemcpyHostToDevice));

  HANDLE_ERROR(
      hipMemcpy(d_kernel_, _kernel, kernelInByte, hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(d_image_, _input, inputInByte, hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(d_initial_, d_image_, inputInByte, hipMemcpyDeviceToDevice));

  // convolve(input) with kernel1 -> psiBlurred
  convolution3DfftCUDAInPlace_core(d_image_, _input_dims, d_kernel_,
                                   _kernel_dims, gpu_device);

  // computeQuotient(psiBlurred,input)
  size_t items_per_block = 128;
  size_t items_per_grid = inputSize / items_per_block;

  dim3 threads(items_per_block);
  dim3 blocks(items_per_grid);
  fit_2Dblocks_to_threads_for_device(threads, blocks, gpu_device);

  // performs d_initial_[:]=d_image_[:]/d_initial_[:]
  // TODO: should the weights be updated?
  device_divide << <blocks, threads>>> (d_initial_, d_image_, inputSize);

  // convolve(psiBlurred) with kernel2 -> integral
  HANDLE_ERROR(hipMemcpy(d_kernel_, &kernel2_[0], kernelInByte,
                          hipMemcpyHostToDevice));
  convolution3DfftCUDAInPlace_core(d_image_, _input_dims, d_kernel_,
                                   _kernel_dims, gpu_device);
  // computeFinalValues(input,integral,weights)
  device_finalValues_tikhonov << <blocks, threads>>>
      (d_initial_, d_image_, d_weights_, .0001f, .2f, inputSize);

  HANDLE_ERROR(
      hipMemcpy(_output, d_initial_, inputInByte, hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipFree(d_image_));
  HANDLE_ERROR(hipFree(d_initial_));
  HANDLE_ERROR(hipFree(d_kernel_));
  HANDLE_ERROR(hipFree(d_weights_));

  delete kernel2_;
  delete weights_;
}
