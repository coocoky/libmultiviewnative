#include "hip/hip_runtime.h"
#define __MULTIVIEWNATIVE_CU__
// ------- C++ ----------
#include <iostream>
#include <cmath>
#include <vector>
#include <algorithm>
#include <numeric>
#include <functional>

// ------- CUDA ----------
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"

// ------- Project ----------
#include "multiviewnative.h"
#include "cuda_helpers.cuh"
#include "cuda_kernels.cuh"
#include "gpu_convolve.cuh"
#include "cufft_utils.cuh"
#include "gpu_deconvolve_methods.cuh"

#include "padd_utils.h"
#include "image_stack_utils.h"


typedef multiviewnative::zero_padd<multiviewnative::image_stack>
    wrap_around_padding;

typedef multiviewnative::no_padd<multiviewnative::image_stack>
    as_is_padding;

typedef multiviewnative::inplace_3d_transform_on_device<imageType>
    device_transform;

typedef multiviewnative::gpu_convolve<wrap_around_padding, imageType, unsigned>
    device_convolve;

//TODO:
//this is the convolution we wanna use in the end
typedef multiviewnative::gpu_convolve<as_is_padding, imageType, unsigned>
    target_convolve;

/**
   \brief Function to perform an inplace convolution (all inputs will received
   wrap_around_padding)

   \param[in] im 1D array that contains the data image stack
   \param[in] imDim 3D array that contains the shape of the image stack im
   \param[in] kernel 1D array that contains the data kernel stack
   \param[in] kernelDim 3D array that contains the shape of the kernel stack
   kernel
   \param[in] device CUDA device to use (see nvidia-smi for details)

   \return
   \retval

*/
void inplace_gpu_convolution(imageType* im, int* imDim, imageType* kernel,
                             int* kernelDim, int device) {

  using namespace multiviewnative;

  unsigned image_dim[3];
  unsigned kernel_dim[3];
  std::copy(imDim, imDim + 3, &image_dim[0]);
  std::copy(kernelDim, kernelDim + 3, &kernel_dim[0]);

  device_convolve convolver(im, image_dim, kernel, kernel_dim);

  if (device < 0) device = selectDeviceWithHighestComputeCapability();

  convolver.set_device(device);

  convolver.inplace<device_transform>();
}



/**
   \brief dispatch function, this one decides wether to try and do the entire
   computation on the device or not and then dispatches the appropriate call



   \return
   \retval

*/
void inplace_gpu_deconvolve(imageType* psi, workspace input, int device) {

  if (device < 0) device = selectDeviceWithHighestComputeCapability();


  long long device_gmem_byte = getMemDeviceCUDA(device);
  unsigned device_gmem_mb = device_gmem_byte >> 20;

  size_t cufft_workarea = 0;
  hipfftEstimate3d(input.data_[0].image_dims_[0], input.data_[0].image_dims_[1],
                  input.data_[0].image_dims_[2], HIPFFT_R2C, &cufft_workarea);
  HANDLE_LAST_ERROR();
  float cufft_workarea_mb = cufft_workarea/(1024*1024.);
  
  size_t single_stack_in_byte =
      sizeof(imageType) * std::accumulate(input.data_[0].image_dims_,
                                          input.data_[0].image_dims_ + 3, 1.,
                                          std::multiplies<int>());
  float single_stack_in_mb = single_stack_in_byte / (1024*1024.);

  float memory_fft_step_mb = 2*single_stack_in_mb + cufft_workarea_mb;
  float regularisation_step_mb = 3*single_stack_in_mb;
  float min_memory_budget_mb = std::max(regularisation_step_mb,memory_fft_step_mb);

  // decide if the incoming data fills the memory on device too much
  float memory_all_on_device_mb = (4*input.num_views_ + 2)*single_stack_in_mb + cufft_workarea_mb;
  

  // cufft is memory hungry, that is why we only push all stacks to device mem
  // if the total budget does not exceed 1/3 device mem
  bool all_on_device = memory_all_on_device_mb < (device_gmem_mb * .9);
  std::cout << "[lmvn::inplace_gpu_deconvolve] FFT: "
            << memory_all_on_device_mb << " MB (all-on-device), "
	    << min_memory_budget_mb << " MB (min-mem interleaved), "
            << " available on GPU: " << device_gmem_mb
            << " MB ... ";

  if (all_on_device){
    std::cout << "all on device!\n";
    inplace_gpu_deconvolve_iteration_all_on_device<wrap_around_padding, device_transform>(psi, input, device);
    return;
  }

  if(min_memory_budget_mb < device_gmem_mb){
    std::cout << "interleaved!\n";
    inplace_gpu_deconvolve_iteration_interleaved<as_is_padding, 
						 target_convolve,
						 device_transform>(psi, input, device);
    return;
  }

  std::cerr << "[lmvn::inplace_gpu_deconvolve] FFT: Unable to run on GPU due to memory constraints!\n";
  
}

#ifndef LB_MAX_THREADS
#define LB_MAX_THREADS 1024
#endif

#ifndef DIMSIMAGE
static const int dimsImage = 3;
#else
static const int dimsImage = DIMSIMAGE;
#endif

__global__ void __launch_bounds__(LB_MAX_THREADS)
    fftShiftKernel(imageType* kernelCUDA, imageType* kernelPaddedCUDA,
                   unsigned int kernelDim_0, unsigned int kernelDim_1,
                   unsigned int kernelDim_2, unsigned int imDim_0,
                   unsigned int imDim_1, unsigned int imDim_2) {
  long int kernelSize = kernelDim_0 * kernelDim_1 * kernelDim_2;
  long int imageSize = imDim_0 * imDim_1 * imDim_2;

  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  long int x, y, z, aux;
  if (tid < kernelSize) {
    // find coordinates
    z = tid - (tid / kernelDim_2) * kernelDim_2;
    aux = (tid - z) / kernelDim_2;
    y = aux - (aux / kernelDim_1) * kernelDim_1;
    x = (aux - y) / kernelDim_1;

    // center coordinates
    x -= (long int)kernelDim_0 / 2;
    y -= (long int)kernelDim_1 / 2;
    z -= (long int)kernelDim_2 / 2;

    // circular shift if necessary
    if (x < 0) x += imDim_0;
    if (y < 0) y += imDim_1;
    if (z < 0) z += imDim_2;

    // WOW! this is a depth-major format
    // calculate position in padded kernel
    aux = z + imDim_2 * (y + imDim_1 * x);

    // copy value
    if (aux < imageSize)
      kernelPaddedCUDA[aux] = kernelCUDA[tid];  // for the most part it should
                                                // be a coalescent access in oth
                                                // places
  }
}

//=====================================================================
// WARNING: for cuFFT the fastest running index is z direction!!! so pos = z +
// imDim[2] * (y + imDim[1] * x)
// NOTE: to avoid transferring a large padded kernel, since memcpy is a limiting
// factor
void convolution3DfftCUDAInPlace(imageType* im, int* imDim, imageType* kernel,
                                 int* kernelDim, int devCUDA) {
  imageType* imCUDA = NULL;
  imageType* kernelCUDA = NULL;

  HANDLE_ERROR(hipSetDevice(devCUDA));

  size_t imSize = std::accumulate(imDim, imDim + 3, 1, std::multiplies<int>());
  size_t kernelSize =
      std::accumulate(kernelDim, kernelDim + 3, 1, std::multiplies<int>());

  size_t imSizeFFT = imSize;
  imSizeFFT +=
      2 * imDim[0] * imDim[1];  // size of the R2C transform in cuFFTComplex
  size_t imSizeFFTInByte = imSizeFFT * sizeof(imageType);
  size_t imSizeInByte = imSize * sizeof(imageType);
  size_t kernelSizeInByte = (kernelSize) * sizeof(imageType);
  // allocat ememory in GPU
  HANDLE_ERROR(hipMalloc((void**)&(imCUDA), imSizeFFTInByte));  // a little bit
                                                                 // larger to
                                                                 // allow
                                                                 // in-place FFT
  HANDLE_ERROR(hipMalloc((void**)&(kernelCUDA), kernelSizeInByte));

  HANDLE_ERROR(
      hipMemcpy(kernelCUDA, kernel, kernelSizeInByte, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(imCUDA, im, imSizeInByte, hipMemcpyHostToDevice));

  ///////////////////////////////////////////////////////////////////////
  convolution3DfftCUDAInPlace_core(imCUDA, imDim, kernelCUDA, kernelDim,
                                   devCUDA);
  ///////////////////////////////////////////////////////////////////////

  // copy result to host and overwrite image
  HANDLE_ERROR(hipMemcpy(im, imCUDA, sizeof(imageType) * imSize,
                          hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipFree(imCUDA));
  HANDLE_ERROR(hipFree(kernelCUDA));
}

//=====================================================================
// WARNING: for cuFFT the fastest running index is z direction!!! so pos = z +
// imDim[2] * (y + imDim[1] * x)
// NOTE: to avoid transferring a large padded kernel, since memcpy is a limiting
// factor
void convolution3DfftCUDAInPlace_core(imageType* _d_imCUDA, int* imDim,
                                      imageType* _d_kernelCUDA, int* kernelDim,
                                      int devCUDA) {
  hipfftHandle fftPlanFwd, fftPlanInv;
  imageType* kernelPaddedCUDA = NULL;

  size_t imSize = 1;
  size_t kernelSize = 1;
  for (int ii = 0; ii < dimsImage; ii++) {
    imSize *= (imDim[ii]);
    kernelSize *= (kernelDim[ii]);
  }

  size_t imSizeFFT = imSize;
  imSizeFFT += 2 * imDim[0] * imDim[1];
  size_t imSizeFFTInByte = imSizeFFT * sizeof(imageType);

  HANDLE_ERROR(hipMalloc((void**)&(kernelPaddedCUDA), imSizeFFTInByte));
  HANDLE_ERROR(hipMemset(kernelPaddedCUDA, 0, imSizeFFTInByte));

  size_t max_threads_on_device = getMaxNThreadsOfDevice(devCUDA);
  size_t max_blocks_in_x =
      getMaxNBlocksOfDevice(devCUDA, 0);  // we are using dim1 blocks only

  int numThreads = std::min(max_threads_on_device, kernelSize);
  size_t numBlocksFromImage = (kernelSize + numThreads - 1) / (numThreads);
  int numBlocks = std::min(max_blocks_in_x, numBlocksFromImage);

  fftShiftKernel << <numBlocks, numThreads>>>
      (_d_kernelCUDA, kernelPaddedCUDA, kernelDim[0], kernelDim[1],
       kernelDim[2], imDim[0], imDim[1], imDim[2]);
  HANDLE_ERROR_KERNEL;

  // make sure GPU finishes
  HANDLE_ERROR(hipDeviceSynchronize());

  hipfftPlan3d(&fftPlanFwd, imDim[0], imDim[1], imDim[2], HIPFFT_R2C);
  HANDLE_ERROR_KERNEL;
  cufftSetCompatibilityMode(fftPlanFwd, CUFFT_COMPATIBILITY_NATIVE);
  HANDLE_ERROR_KERNEL;  // for highest performance since we do not need FFTW
                        // compatibility

  // inPlace FFT for image and kernel
  hipfftExecR2C(fftPlanFwd, _d_imCUDA, (hipfftComplex*)_d_imCUDA);
  HANDLE_ERROR_KERNEL;
  hipfftExecR2C(fftPlanFwd, kernelPaddedCUDA, (hipfftComplex*)kernelPaddedCUDA);
  HANDLE_ERROR_KERNEL;

  size_t halfImSizeFFT = imSizeFFT / 2;
  numThreads = std::min(max_threads_on_device, halfImSizeFFT);
  numBlocksFromImage = (halfImSizeFFT + numThreads - 1) / (numThreads);
  numBlocks = std::min(max_blocks_in_x, numBlocksFromImage);

  // convolve
  float scale = 1.0f / float(imSize);
  modulateAndNormalize_kernel << <numBlocks, numThreads>>>
      ((hipfftComplex*)_d_imCUDA, (hipfftComplex*)kernelPaddedCUDA, halfImSizeFFT,
       scale);
  HANDLE_ERROR_KERNEL;  // last parameter is the size of the FFT

  // inverse FFT of image only
  hipfftPlan3d(&fftPlanInv, imDim[0], imDim[1], imDim[2], HIPFFT_C2R);
  HANDLE_ERROR_KERNEL;
  cufftSetCompatibilityMode(fftPlanInv, CUFFT_COMPATIBILITY_NATIVE);
  HANDLE_ERROR_KERNEL;
  hipfftExecC2R(fftPlanInv, (hipfftComplex*)_d_imCUDA, _d_imCUDA);
  HANDLE_ERROR_KERNEL;

  // release memory
  HANDLE_ERROR(hipFree(kernelPaddedCUDA));
  (hipfftDestroy(fftPlanFwd));
  HANDLE_ERROR_KERNEL;
  (hipfftDestroy(fftPlanInv));
  HANDLE_ERROR_KERNEL;
}

void compute_quotient(imageType* _input, imageType* _output, size_t _size,
                      int _device) {

  imageType* d_input = 0;
  imageType* d_output = 0;

  const size_t sizeInByte = _size * sizeof(imageType);
  HANDLE_ERROR(hipSetDevice(_device));

  HANDLE_ERROR(hipMalloc((void**)&d_input, sizeInByte));
  HANDLE_ERROR(hipMalloc((void**)&d_output, sizeInByte));

  HANDLE_ERROR(hipMemcpy(d_input, _input, sizeInByte, hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(d_output, _output, sizeInByte, hipMemcpyHostToDevice));

  size_t items_per_block = 128;
  size_t items_per_grid = _size / items_per_block;

  dim3 threads(items_per_block);
  dim3 blocks(items_per_grid);

  fit_2Dblocks_to_threads_for_device(threads, blocks, _device);

  // performs d_input_[:]=d_output_[:]/d_input_[:]
  device_divide << <blocks, threads>>> (d_input, d_output, (unsigned int)_size);

  HANDLE_ERROR(
      hipMemcpy(_output, d_output, sizeInByte, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipFree(d_input));
  HANDLE_ERROR(hipFree(d_output));
}

void compute_final_values(imageType* _image, imageType* _integral,
                          imageType* _weight, size_t _size, float _minValue,
                          double _lambda, int _device) {

  imageType* d_integral = 0;
  imageType* d_weight = 0;
  imageType* d_image = 0;

  const size_t sizeInByte = _size * sizeof(imageType);

  HANDLE_ERROR(hipMalloc((void**)&d_integral, sizeInByte));
  HANDLE_ERROR(hipMalloc((void**)&d_weight, sizeInByte));
  HANDLE_ERROR(hipMalloc((void**)&d_image, sizeInByte));

  HANDLE_ERROR(
      hipMemcpy(d_integral, _integral, sizeInByte, hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(d_weight, _weight, sizeInByte, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(d_image, _image, sizeInByte, hipMemcpyHostToDevice));

  size_t items_per_block = 128;
  size_t items_per_grid = _size / items_per_block;

  dim3 threads(items_per_block);
  dim3 blocks(items_per_grid);
  fit_2Dblocks_to_threads_for_device(threads, blocks, _device);

  if (_lambda > 0.)
    device_regularized_final_values << <blocks, threads>>>
        (d_image, d_integral, d_weight, _lambda, _minValue, _size);
  else
    device_final_values << <blocks, threads>>>
        (d_image, d_integral, d_weight, _minValue, _size);

  HANDLE_ERROR(hipMemcpy(_image, d_image, sizeInByte, hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipFree(d_integral));
  HANDLE_ERROR(hipFree(d_weight));
  HANDLE_ERROR(hipFree(d_image));
}

void iterate_fft_plain(imageType* _input, imageType* _kernel,
                       imageType* _output, int* _input_dims, int* _kernel_dims,
                       int _device) {

  size_t inputSize = std::accumulate(
      &_input_dims[0], &_input_dims[0] + dimsImage, 1, multiplies<int>());
  size_t kernelSize = std::accumulate(
      &_kernel_dims[0], &_kernel_dims[0] + dimsImage, 1, multiplies<int>());

  size_t inputInByte = inputSize * sizeof(imageType);
  size_t kernelInByte = kernelSize * sizeof(imageType);

  imageType* input = 0;
  imageType* kernel1 = 0;
  imageType* weights = 0;
  imageType* kernel2 = 0;
  HANDLE_ERROR(
      hipHostMalloc((void**)&(kernel2), kernelInByte, hipHostMallocDefault));
  HANDLE_ERROR(
      hipHostMalloc((void**)&(weights), inputInByte, hipHostMallocDefault));
  HANDLE_ERROR(
      hipHostMalloc((void**)&(kernel1), kernelInByte, hipHostMallocDefault));
  HANDLE_ERROR(
      hipHostMalloc((void**)&(input), inputInByte, hipHostMallocDefault));

  std::fill(&kernel2[0], &kernel2[0] + kernelSize, .1f);
  std::fill(&weights[0], &weights[0] + inputSize, 1.f);
  std::copy(&_kernel[0], &_kernel[0] + kernelSize, &kernel1[0]);
  std::copy(&_input[0], &_input[0] + inputSize, &input[0]);

  //////////////////////////////////////////////////////////////////////////////////////////
  //
  // Entering Loop here
  //
  imageType* d_image_ = 0;
  imageType* d_initial_ = 0;
  imageType* d_kernel_ = 0;
  imageType* d_weights_ = 0;

  size_t imSizeFFT =
      inputSize +
      2 * _input_dims[0] *
          _input_dims[1];  // size of the R2C transform in cuFFTComplex
  size_t imSizeFFTInByte = imSizeFFT * sizeof(imageType);
  int gpu_device = selectDeviceWithHighestComputeCapability();

  HANDLE_ERROR(hipMalloc((void**)&(d_image_), imSizeFFTInByte));  // a little
                                                                   // bit larger
                                                                   // to allow
                                                                   // in-place
                                                                   // FFT
  HANDLE_ERROR(hipMalloc((void**)&(d_initial_), inputInByte));
  HANDLE_ERROR(hipMalloc((void**)&(d_weights_), inputInByte));
  HANDLE_ERROR(hipMalloc((void**)&(d_kernel_), kernelInByte));
  hipStream_t initial_stream, weights_stream;
  HANDLE_ERROR(hipStreamCreate(&initial_stream));
  HANDLE_ERROR(hipStreamCreate(&weights_stream));

  // TODO: should the weights be updated from device_divide (unclear in the java
  // application)?
  HANDLE_ERROR(hipMemcpyAsync(d_weights_, weights, inputInByte,
                               hipMemcpyHostToDevice, weights_stream));
  HANDLE_ERROR(hipMemcpyAsync(d_initial_, input, inputInByte,
                               hipMemcpyHostToDevice, initial_stream));

  HANDLE_ERROR(
      hipMemcpy(d_kernel_, kernel1, kernelInByte, hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(d_image_, input, inputInByte, hipMemcpyHostToDevice));

  // convolve(input) with kernel1 -> psiBlurred
  convolution3DfftCUDAInPlace_core(d_image_, _input_dims, d_kernel_,
                                   _kernel_dims, gpu_device);

  // computeQuotient(psiBlurred,input)
  size_t items_per_block = 128;
  size_t items_per_grid = inputSize / items_per_block;

  dim3 threads(items_per_block);
  dim3 blocks(items_per_grid);
  fit_2Dblocks_to_threads_for_device(threads, blocks, gpu_device);

  // performs d_initial_[:]=d_image_[:]/d_initial_[:]
  // TODO: should the weights be updated?
  device_divide << <blocks, threads, 0, initial_stream>>>
      (d_initial_, d_image_, inputSize);

  // convolve(psiBlurred) with kernel2 -> integral
  HANDLE_ERROR(
      hipMemcpy(d_kernel_, kernel2, kernelInByte, hipMemcpyHostToDevice));
  convolution3DfftCUDAInPlace_core(d_image_, _input_dims, d_kernel_,
                                   _kernel_dims, gpu_device);
  // computeFinalValues(input,integral,weights)
  device_final_values << <blocks, threads, 0, weights_stream>>>
      (d_initial_, d_image_, d_weights_, .0001f, inputSize);

  HANDLE_ERROR(hipMemcpyAsync(_output, d_initial_, inputInByte,
                               hipMemcpyDeviceToHost, weights_stream));
  HANDLE_ERROR(hipStreamSynchronize(weights_stream));

  HANDLE_ERROR(hipFree(d_image_));
  HANDLE_ERROR(hipFree(d_initial_));
  HANDLE_ERROR(hipFree(d_kernel_));
  HANDLE_ERROR(hipFree(d_weights_));

  HANDLE_ERROR(hipHostFree(kernel2));
  HANDLE_ERROR(hipHostFree(weights));
  HANDLE_ERROR(hipHostFree(kernel1));
  HANDLE_ERROR(hipHostFree(input));
  HANDLE_ERROR(hipStreamDestroy(initial_stream));
  HANDLE_ERROR(hipStreamDestroy(weights_stream));
}

void iterate_fft_tikhonov(imageType* _input, imageType* _kernel,
                          imageType* _output, int* _input_dims,
                          int* _kernel_dims, size_t _size, float _minValue,
                          double _lambda, int _device) {

  size_t inputSize = std::accumulate(
      &_input_dims[0], &_input_dims[0] + dimsImage, 1, multiplies<int>());
  size_t kernelSize = std::accumulate(
      &_kernel_dims[0], &_kernel_dims[0] + dimsImage, 1, multiplies<int>());

  size_t inputInByte = inputSize * sizeof(imageType);
  size_t kernelInByte = kernelSize * sizeof(imageType);

  std::vector<imageType>* kernel2_ = new std::vector<imageType>(kernelSize);
  std::vector<imageType>* weights_ = new std::vector<imageType>(inputSize);
  std::fill(kernel2_->begin(), kernel2_->end(), .1f);
  std::fill(weights_->begin(), weights_->end(), 1.f);

  //////////////////////////////////////////////////////////////////////////////////////////
  //
  // Entering Loop here
  //
  imageType* d_image_ = 0;
  imageType* d_initial_ = 0;
  imageType* d_kernel_ = 0;
  imageType* d_weights_ = 0;

  size_t imSizeFFT =
      inputSize +
      2 * _input_dims[0] *
          _input_dims[1];  // size of the R2C transform in cuFFTComplex
  size_t imSizeFFTInByte = imSizeFFT * sizeof(imageType);
  int gpu_device = selectDeviceWithHighestComputeCapability();

  HANDLE_ERROR(hipMalloc((void**)&(d_image_), imSizeFFTInByte));  // a little
                                                                   // bit larger
                                                                   // to allow
                                                                   // in-place
                                                                   // FFT
  HANDLE_ERROR(hipMalloc((void**)&(d_initial_), inputInByte));
  HANDLE_ERROR(hipMalloc((void**)&(d_weights_), inputInByte));
  HANDLE_ERROR(hipMalloc((void**)&(d_kernel_), kernelInByte));
  HANDLE_ERROR(hipMemcpy(d_weights_, &weights_[0], inputInByte,
                          hipMemcpyHostToDevice));

  HANDLE_ERROR(
      hipMemcpy(d_kernel_, _kernel, kernelInByte, hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(d_image_, _input, inputInByte, hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(d_initial_, d_image_, inputInByte, hipMemcpyDeviceToDevice));

  // convolve(input) with kernel1 -> psiBlurred
  convolution3DfftCUDAInPlace_core(d_image_, _input_dims, d_kernel_,
                                   _kernel_dims, gpu_device);

  // computeQuotient(psiBlurred,input)
  size_t items_per_block = 128;
  size_t items_per_grid = inputSize / items_per_block;

  dim3 threads(items_per_block);
  dim3 blocks(items_per_grid);
  fit_2Dblocks_to_threads_for_device(threads, blocks, gpu_device);

  // performs d_initial_[:]=d_image_[:]/d_initial_[:]
  // TODO: should the weights be updated?
  device_divide << <blocks, threads>>> (d_initial_, d_image_, inputSize);

  // convolve(psiBlurred) with kernel2 -> integral
  HANDLE_ERROR(hipMemcpy(d_kernel_, &kernel2_[0], kernelInByte,
                          hipMemcpyHostToDevice));
  convolution3DfftCUDAInPlace_core(d_image_, _input_dims, d_kernel_,
                                   _kernel_dims, gpu_device);
  // computeFinalValues(input,integral,weights)
  device_finalValues_tikhonov << <blocks, threads>>>
      (d_initial_, d_image_, d_weights_, .0001f, .2f, inputSize);

  HANDLE_ERROR(
      hipMemcpy(_output, d_initial_, inputInByte, hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipFree(d_image_));
  HANDLE_ERROR(hipFree(d_initial_));
  HANDLE_ERROR(hipFree(d_kernel_));
  HANDLE_ERROR(hipFree(d_weights_));

  delete kernel2_;
  delete weights_;
}
