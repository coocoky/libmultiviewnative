#include "hip/hip_runtime.h"
#define BOOST_TEST_DYN_LINK 
#define BOOST_TEST_MODULE CUDA_MEMORY_SUITE
#include "boost/test/unit_test.hpp"
#include "test_fixtures.hpp"
#include "cuda_memory.cuh"

template <typename T>
__global__ void add_1(T* _container, unsigned _size){

  unsigned global = threadIdx.x + blockIdx.x*blockDim.x;

  if(global < _size)
    _container[global] += 1.f;
  
}

typedef multiviewnative::stack_on_device<multiviewnative::image_stack> default_stack_on_device;
typedef multiviewnative::stack_on_device<multiviewnative::image_stack, multiviewnative::asynch> asynch_stack_on_device;
typedef multiviewnative::stack_on_device<multiviewnative::image_stack, multiviewnative::synch>  synch_stack_on_device;

BOOST_FIXTURE_TEST_SUITE( constructor_suite, multiviewnative::default_3D_fixture )
   
BOOST_AUTO_TEST_CASE( instantiate )
{
  synch_stack_on_device nullary;
  hipError_t err = hipGetLastError();
  BOOST_CHECK_EQUAL(err, hipSuccess);
}

BOOST_AUTO_TEST_CASE( instantiate_from_number )
{
  synch_stack_on_device any_size(256);
  BOOST_CHECK(any_size.device_stack_ptr_ != 0);
  hipError_t err = hipGetLastError();
  BOOST_CHECK_EQUAL(err, hipSuccess);
}


BOOST_AUTO_TEST_CASE( by_from_stack )
{
  synch_stack_on_device simple(image_);
  BOOST_CHECK_MESSAGE(simple.device_stack_ptr_ != 0, "stack_on_device has no device memory loaded");
  BOOST_CHECK_EQUAL(simple.size_in_byte_, image_.num_elements()*sizeof(synch_stack_on_device::value_type));
  hipError_t err = hipGetLastError();
  BOOST_CHECK_EQUAL(err, hipSuccess);
}


BOOST_AUTO_TEST_CASE( by_assigment )
{
  default_stack_on_device simple(image_);
  hipError_t err = hipGetLastError();
  BOOST_CHECK_EQUAL(err, hipSuccess);

  {
    default_stack_on_device simple2 = simple;
  }
  
  err = hipGetLastError();
  BOOST_CHECK_EQUAL(err, hipSuccess);
}


BOOST_AUTO_TEST_CASE( by_assigment_from_stack )
{
  default_stack_on_device simple = image_;
  BOOST_CHECK_MESSAGE(simple.device_stack_ptr_ != 0, "stack_on_device has no device memory loaded");
  BOOST_CHECK_EQUAL(simple.size_in_byte_, image_.num_elements()*sizeof(default_stack_on_device::value_type));
}


BOOST_AUTO_TEST_SUITE_END()

BOOST_FIXTURE_TEST_SUITE( offload_suite, multiviewnative::default_3D_fixture )
   
BOOST_AUTO_TEST_CASE( instantiate_add_1_synched )
{
  using namespace multiviewnative;

  unsigned sum_original = std::accumulate(image_.data(), image_.data() + image_.num_elements(), 0.);
  synch_stack_on_device simple(image_);
  
  simple.push_to_device(image_);

  dim3 blocks = image_.shape()[2];
  dim3 threads = image_.shape()[1]*image_.shape()[0];
  
  add_1<<<blocks,threads>>>(simple.device_stack_ptr_, image_.num_elements());

  simple.pull_from_device(image_);

  unsigned sum = std::accumulate(image_.data(), image_.data() + image_.num_elements(), 0.);
  
  BOOST_CHECK_NE(sum_original, sum);
  BOOST_CHECK_EQUAL(sum_original+image_.num_elements(), sum);
}

BOOST_AUTO_TEST_CASE( instantiate_add_1_asynched )
{
  using namespace multiviewnative;
  hipStream_t tstream;
  hipStreamCreate(&tstream);

  unsigned sum_original = std::accumulate(image_.data(), image_.data() + image_.num_elements(), 0.);
  asynch_stack_on_device simple(image_, image_.num_elements(), &tstream);
  
  dim3 blocks = image_.shape()[2];
  dim3 threads = image_.shape()[1]*image_.shape()[0];
  
  add_1<<<blocks,threads, 0 , tstream>>>(simple.device_stack_ptr_, image_.num_elements());

  simple.pull_from_device(image_,&tstream);

  hipStreamSynchronize(tstream);
  unsigned sum = std::accumulate(image_.data(), image_.data() + image_.num_elements(), 0.);
  
  BOOST_CHECK_NE(sum_original, sum);
  BOOST_CHECK_EQUAL(sum_original+image_.num_elements(), sum);
  hipStreamDestroy(tstream);
}

BOOST_AUTO_TEST_SUITE_END()


















